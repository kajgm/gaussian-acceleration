#include "hip/hip_runtime.h"
#include "libwb/wb.h"
#include "my_timer.h"
#include <stdio.h>

#define wbCheck(stmt)                                                \
  do                                                                 \
  {                                                                  \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
      return -1;                                                     \
    }                                                                \
  } while (0)

#define SIZE 1024
#define RAND_SEED 123

#define TILE_SIZE 32

///////////////////////////////////////////////////////
//@@ INSERT YOUR CODE HERE
__global__ void gaussKernel(float *A, float *B, float *X)
{
}

int main(int argc, char *argv[])
{

  float *A_hw_host;
  float *B_hw_host;
  float *X_hw_host;

  float *A_hw_device;
  float *B_hw_device;
  float *X_hw_device;

  float *A_sw;
  float *B_sw;
  float *X_sw;

  srand(RAND_SEED);

  /* Initialize A, B, and X */
  initialize_inputs(A_sw, B_sw, X_sw, A_hw_host, B_hw_host, X_hw_host);

  gauss_sw(A_sw, B_sw, X_sw);

  // Allocate cuda memory for device input and ouput image data
  hipHostAlloc((void **)&A_hw_device,
                SIZE * SIZE * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void **)&B_hw_device,
                SIZE * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void **)&X_hw_device,
                SIZE * sizeof(float), hipHostMallocDefault);

  // Start timer
  timespec timer = tic();

  ////////////////////////////////////////////////
  //@@ INSERT AND UPDATE YOUR CODE HERE
  hipMalloc((void **)&A_hw_device,
             SIZE * SIZE * sizeof(float));
  hipMalloc((void **)&B_hw_device,
             SIZE * sizeof(float));
  hipMalloc((void **)&X_hw_device,
             SIZE * sizeof(float));

  // Transfer data from CPU to GPU
  hipMemcpy(A_hw_device, A_hw_host,
             SIZE * SIZE * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(B_hw_device, B_hw_host,
             SIZE * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(X_hw_device, X_hw_host,
             SIZE * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 DimGrid((SIZE + TILE_SIZE - 1) / TILE_SIZE, (SIZE + TILE_SIZE - 1) / TILE_SIZE);
  dim3 DimBlock(TILE_SIZE, TILE_SIZE);

  // Call your GPU kernel 10 times
  for (int i = 0; i < 10; i++)
    gaussKernel<<<DimGrid, DimBlock>>>(A_hw_device, B_hw_device, X_hw_device);

  // Transfer data from GPU to CPU
  hipMemcpy(A_hw_host, A_hw_device,
             SIZE * SIZE * sizeof(float),
             hipMemcpyDeviceToHost);
  hipMemcpy(B_hw_host, B_hw_device,
             SIZE * sizeof(float),
             hipMemcpyDeviceToHost);
  hipMemcpy(X_hw_host, X_hw_device,
             SIZE * sizeof(float),
             hipMemcpyDeviceToHost);
  ///////////////////////////////////////////////////////

  // Stop and print timer
  toc(&timer, "GPU execution time (including data transfer) in seconds");

  // Check the correctness of your solution
  // wbSolution(args, outputImage);

  for (int i = 0; i < SIZE; i++)
  {
    for (int j = 0; j < SIZE; j++)
    {
      if (abs(A_hw_host[i * SIZE + j] - A_sw[i * SIZE + j]) / A_sw[i * SIZE + j] > 0.01)
      {
        printf("Incorrect A value at (%d, %d): A_sw = %f, A_hw = %f\n", i, j, A_sw[i * SIZE + j], A_hw_host[i * SIZE + j]);
        return -1;
      }
    }

    if (abs(B_hw_host[i] - B_sw[i]) / B_sw[i] > 0.01)
    {
      printf("Incorrect B value at (%d, %d): B_sw = %f, B_hw = %f\n", i, j, B_sw[i], B_hw_host[i]);
      return -1;
    }
    if (abs(X_hw_host[i] - X_sw[i]) / X_sw[i] > 0.01)
    {
      printf("Incorrect X value at (%d, %d): X_sw = %f, X_hw = %f\n", i, j, X_sw[i], X_hw_host[i]);
      return -1;
    }
  }
  printf("Correct output Matrix!\n");

  hipHostFree(A_hw_device);
  hipHostFree(B_hw_device);
  hipHostFree(X_hw_device);

  hipFree(A_hw_device);
  hipFree(B_hw_device);
  hipFree(X_hw_device);

  return 0;
}
